#include "hip/hip_runtime.h"
#include "quantum.cuh"

/* QUBITS FUNCTIONS */

__host__ simulator initSimulatorDevice(int size) {
	simulator d_sim;
	int qbit_size = sizeof(qubit) * size;
	int array_size = sizeof(int) * size;
	
	hipMalloc((void**) &d_sim, sizeof(simulator));
	hipMalloc((void**) &d_sim.size, sizeof(int));
	hipMalloc((void**) &d_sim.q, qbit_size);
	hipMalloc((void**) &d_sim.mesure, array_size);
	hipMalloc((void**) &d_sim.target, array_size);
	for(int i = 0; i < size; i++)
		hipMalloc((void**) &d_sim.control[i], array_size);

	//d_sim.size = size;
	printf("Device criado com sucesso.\n");
	return d_sim;
}

__host__ simulator initSimulatorHost(int size) {
	simulator sim;
	sim.size = size;
	sim.q = (qubit*)calloc(size, sizeof(qubit));
	if (sim.q == NULL) exit(-1);
	for (int i = 0; i < sim.size; i++)
		sim.q[i].amplitude[0].real = (float) 1;
	
	sim.mesure = (int*)calloc(size, sizeof(int));
	if (sim.mesure == NULL) exit(-1);
	
	sim.target = (int*)calloc(size, sizeof(int));
	if (sim.target == NULL) exit(-1);
	
	for (int i = 0; i < size; i++) {
		sim.control[i] = (int*)calloc(size, sizeof(int));
		if (sim.control[i] == NULL) exit(-1);
	}
	
	printf("Host criado com sucesso.\n");
	return sim;
}

__host__ void cpyToDevice(simulator ori, simulator dest) {

	int qbit_size = sizeof(qubit) * ori.size;
	int array_size = sizeof(int) * ori.size;

	dest.size = ori.size;
	hipMemcpy(dest.q, ori.q, qbit_size, hipMemcpyHostToDevice);
	hipMemcpy(dest.mesure, ori.mesure, array_size, hipMemcpyHostToDevice);
	hipMemcpy(dest.target, ori.target, array_size, hipMemcpyHostToDevice);
	//hipMemcpy(dest.size, ori.size, sizeof(int), hipMemcpyHostToDevice);
	for (int i = 0; i < ori.size; i++)
		hipMemcpy(dest.control[i], ori.control[i], array_size, hipMemcpyHostToDevice);

	printf("Copiado para device.\n");
}

__host__ void cpyToHost(simulator ori, simulator dest) {

	int qbit_size = sizeof(qubit) * dest.size;
	int array_size = sizeof(int) * dest.size;

	hipMemcpy(dest.q,      ori.q,      qbit_size,  hipMemcpyDeviceToHost);
	hipMemcpy(dest.mesure, ori.mesure, array_size, hipMemcpyDeviceToHost);
	hipMemcpy(dest.target, ori.target, array_size, hipMemcpyDeviceToHost);
	for (int i = 0; i < dest.size; i++)
		hipMemcpy(dest.control[i], ori.control[i], array_size, hipMemcpyDeviceToHost);

	printf("Copiado para host.\n");
}

__host__ void freeSimulatorDevice(simulator d_simu) {
	hipFree(d_simu.q);
	hipFree(d_simu.mesure);
	hipFree(d_simu.target);
	for(int i = 0; i < 2; i++)
		hipFree(d_simu.control[i]);
	hipFree(d_simu.control);
	printf("Device liberado.\n");
}

__host__ void freeSimulatorHost(simulator simu) {
	free(simu.q);
	free(simu.mesure);
	free(simu.target);
	for(int i = 0; i < 2; i++)
		free(simu.control[i]);
	printf("Host liberado.\n");
}

/* OTHER FUNCTIONS*/

complex complexProduct(complex a, complex b) {
	complex c;
	c.real = (a.real * b.real) - (a.imag * b.imag);
	c.imag = (a.real * b.imag) + (a.imag * b.real);

	return c;
}

__host__ void printQubit(simulator sim, int mesure) {
	for (int i = 0; i < sim.size; i++) {
		printf("Qubit [%d]:\n", i);
		for (int j = 0; j < 2; j++) {
			printf("|%d>: (%.2f + %.2fi)\t", j, sim.q[i].amplitude[j].real, sim.q[i].amplitude[j].imag);
		}
		printf("\n\n");
	}
	if(mesure)
		for (int i = 0; i < sim.size; i++) {
			printf("Qubit [%d]: %d\n", i, sim.mesure[i]);
		}
}

__global__ void mesureQubit(qubit* q, int* mesure_vector, float percentage) {
	int index = threadIdx.x;

	float sum = .0f, prev = .0f;
	int mesure_result = 0;
	for (int i = 0; i < q[index].size; i++)
		mesure_vector[i] = i;
	/*
	for (int i = 0; i < q[index].size; i++) {
		float a = q[index].amplitude[1].real * q[index].amplitude[1].real;
		float b = q[index].amplitude[1].imag * q[index].amplitude[1].imag;

		q[index].amplitude[1].imag = percentage;
		q[index].amplitude[1].real = percentage;
		
		sum += (a + b);
		if (prev <= percentage && percentage <= sum) {
			mesure_vector[index] = i;
		}
		prev = sum;
	}
	*/
	
		
}

/* QUANTUM GATES */

// Toffoli e cnot precisam representar estados emaranhados
__global__ void toffoliGate(qubit* d_q, int* t, int* c1, int* c2) {
	int index = threadIdx.x;

	// A aplcação da porta só pode ser efetuada se os vetores possuirem o mesmo tamanho
	if ((sizeof(t) / sizeof(t[0])) == (sizeof(c1) / sizeof(c1[0])) &&
		(sizeof(t) / sizeof(t[0])) == (sizeof(c2) / sizeof(c2[0])))
	// Se os qubit c1 e c2 possuirem amplitudes do vetor |1> diferente de 0, trocar o sinal do qubit em t
	if ((d_q[c1[index]].amplitude[1].real != .0f || d_q[c1[index]].amplitude[1].imag != .0f) &&
		(d_q[c2[index]].amplitude[1].real != .0f || d_q[c2[index]].amplitude[1].imag != .0f)) {
		complex aux = d_q[t[index]].amplitude[0];
		d_q[t[index]].amplitude[0] = d_q[t[index]].amplitude[1];
		d_q[t[index]].amplitude[1] = aux;
	}
}

__global__ void cnotGate(qubit* d_q, int* t, int* ctrl) {
	// t é um ponteiro para vetor de qubits a serem afetados pela porta cnotGate
	// ctrl é um ponteiro para vetor de qubits de controle
	int index = threadIdx.x;
	// Se o qubit ctrl possuir amplitude do vetor |1> diferente de 0, trocar o sinal do qubit em t
	if((sizeof(t) / sizeof(t[0])) == (sizeof(ctrl) / sizeof(ctrl[0])))
	if (d_q[ctrl[index]].amplitude[1].real != .0f || d_q[ctrl[index]].amplitude[1].imag != .0f) {
		complex aux = d_q[t[index]].amplitude[0];
		d_q[t[index]].amplitude[0] = d_q[t[index]].amplitude[1];
		d_q[t[index]].amplitude[1] = aux;
	}
}

__global__ void notGate(qubit* d_q) {
	int index = threadIdx.x;

	complex aux = d_q[index].amplitude[0];
	d_q[index].amplitude[0] = d_q[index].amplitude[1];
	d_q[index].amplitude[1] = aux;
}

__global__ void hadamardGate(qubit* d_q) {
	int index = threadIdx.x;

	float ampH = 0.70710678118;
	complex alpha, beta;

	float a1 = d_q[index].amplitude[0].real;
	float a2 = d_q[index].amplitude[0].imag;
	float b1 = d_q[index].amplitude[1].real;
	float b2 = d_q[index].amplitude[1].imag;

	alpha.real = (a1 + b1) * ampH;
	alpha.imag = (a2 + b2) * ampH;

	beta.real = (a1 - b1) * ampH;
	beta.imag = (a2 - b2) * ampH;

	d_q[index].amplitude[0] = alpha;
	d_q[index].amplitude[1] = beta;
}

__global__ void phaseGate(qubit* d_q) {
	int index = threadIdx.x;
	float b = -d_q[index].amplitude[1].imag;
	float c = d_q[index].amplitude[1].real;
	d_q[index].amplitude[1].real = b;
	d_q[index].amplitude[1].imag = c;
}

// As funções _T aplicam o resultado da operação nos qbits indicados em target

__global__ void notGate_T(qubit* d_q, int *target) {
	int index = threadIdx.x;
	complex aux = d_q[target[index]].amplitude[0];
	d_q[target[index]].amplitude[0] = d_q[target[index]].amplitude[1];
	d_q[target[index]].amplitude[1] = aux;
}

__global__ void hadamardGate_T(qubit* d_q, int *target) {
	int index = threadIdx.x;

	float ampH = 0.70710678118;
	complex alpha, beta;

	float a1 = d_q[target[index]].amplitude[0].real;
	float a2 = d_q[target[index]].amplitude[0].imag;
	float b1 = d_q[target[index]].amplitude[1].real;
	float b2 = d_q[target[index]].amplitude[1].imag;

	alpha.real = (a1 + b1) * ampH;
	alpha.imag = (a2 + b2) * ampH;

	beta.real = (a1 - b1) * ampH;
	beta.imag = (a2 - b2) * ampH;

	d_q[target[index]].amplitude[0] = alpha;
	d_q[target[index]].amplitude[1] = beta;
}

__global__ void phaseGate_T(qubit* q, int *target) {
	int index = threadIdx.x;
	float b = -q[target[index]].amplitude[1].imag;
	float c = q[target[index]].amplitude[1].real;
	q[target[index]].amplitude[1].real = b;
	q[target[index]].amplitude[1].imag = c;
}

// As funções '_All' aplicam o resultado em todos os qbits

__global__ void notGate_All(simulator sim){
	int index = threadIdx.x;
	if(index < sim.size){
		complex aux = sim.q[index].amplitude[0];
		sim.q[index].amplitude[0] = sim.q[index].amplitude[1];
		sim.q[index].amplitude[1] = aux;
	}
}

__global__ void hadamardGate_All(simulator sim){
	int index = threadIdx.x;
	if(index < sim.size){
		float ampH = 0.70710678118;
		complex alpha, beta;

		float a1 = sim.q[index].amplitude[0].real;
		float a2 = sim.q[index].amplitude[0].imag;
		float b1 = sim.q[index].amplitude[1].real;
		float b2 = sim.q[index].amplitude[1].imag;

		alpha.real = (a1 + b1) * ampH;
		alpha.imag = (a2 + b2) * ampH;

		beta.real = (a1 - b1) * ampH;
		beta.imag = (a2 - b2) * ampH;

		sim.q[index].amplitude[0] = alpha;
		sim.q[index].amplitude[1] = beta;
	}
}

__global__ void phaseGate_All(simulator sim){
	int index = threadIdx.x;
	if(index < sim.size){
		float b = -sim.q[index].amplitude[1].imag;
		float c = sim.q[index].amplitude[1].real;
		sim.q[index].amplitude[1].real = b;
		sim.q[index].amplitude[1].imag = c;
	}
}
