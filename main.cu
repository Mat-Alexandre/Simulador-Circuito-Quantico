#include "hip/hip_runtime.h"
#include <stdio.h>
#include "quantum.cuh"

__global__
void notGate(struct qubit *Q){
    
}

int main(){
    struct qubit q;
    struct qubit *d_q;

    // Allocating the qubit pointer
    hipMalloc((void **) &d_q, sizeof(struct qubit));

    // Copying the data from q to d_q
    hipMemcpy(d_q, &q, sizeof(struct qubit), hipMemcpyHostToDevice);

    // Executing the notGate on GPU
    notGate<<<1,1>>>(d_q);

    // Copying back the value of d_q to q
    hipMemcpy(&q, d_q, sizeof(struct qubit), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_q);

    return 0;
}