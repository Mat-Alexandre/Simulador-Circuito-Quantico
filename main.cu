#include <stdio.h>
#include "quantum.cuh"

#define N 4
#define T 2

int main() {
	int qbit_size = N * sizeof(qubit);
	int vect_size = T * sizeof(int);
	
	qubit q[N], *d_q;
	
	// Valores para teste
	q[0].amplitude[1].real = 0.0f;
	q[1].amplitude[1].real = 0.0f;
	q[2].amplitude[0].real = 0.0f;
	q[3].amplitude[0].real = 0.0f;

	// Vetores alvo e controles
	int t[T] = { 0, 1 }, c1[T] = { 2, 3 }, c2[T];
	int *d_t, *d_c1, *d_c2;

	// Alocação dos vetores Devices
	hipMalloc((void**) &d_q, qbit_size);
	hipMalloc((void**) &d_t, vect_size);
	hipMalloc((void**)&d_c1, vect_size);

	// Cópia para as variáveis Devices
	hipMemcpy( d_q,  q, qbit_size, hipMemcpyHostToDevice);
	hipMemcpy( d_t,  t, vect_size, hipMemcpyHostToDevice);
	hipMemcpy(d_c1, c1, vect_size, hipMemcpyHostToDevice);
	
	// Imprimindo os resultados iniciais
	printf("-------INICIO-------\n");
	printQubit(q, N);
	printf("Qubit alvo: %d\n", t[0]);
	printf("Qubit controle: %d\n", c1[0]);

	// Aplicação da(s) porta(s)
	cnotGate<<<1, N>>>(d_q, d_t, d_c1);

	// Cópia para as variáveis Host
	hipMemcpy( q, d_q,  qbit_size, hipMemcpyDeviceToHost);
	hipMemcpy( t, d_t,  vect_size, hipMemcpyDeviceToHost);
	hipMemcpy(c1, d_c1, vect_size, hipMemcpyDeviceToHost);

	// Imprimindo os resultados
	printf("------RESULTADO-----\n");
	printQubit(q, N);
	
	// Liberando as variáveis alocadas
	hipFree(d_q);
	hipFree(d_t);
	hipFree(d_c1);

	return 0;
}
